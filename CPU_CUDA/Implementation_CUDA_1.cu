#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <vector>
#include <cmath>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <helper/helper_functions.h>
#include <helper/hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
typedef float2 cplx;
#define RANK 1
#define M_PI 3.14159265358979323846
#define FLOAT_MAX 1
#define MAX_THREADS 512
struct kernelConf
{
    dim3 block;
    dim3 grid;
};
using namespace std;
using namespace cv;
void readHeaderFile(char *filen, int * dims);
int readCflFile(char *file, cplx* data);
int DoDisplayImage2CV(float* dataRaw, float* dataRecons, int xdim, int ydim, int slice, int zipDim, char* filename,int numSlice);
__global__ void DoRSSCUDA(cplx* input, float* out, int N, int x, int y, int bit, int numSlice);
__global__ void DoGetSpecificSliceDataCUDA(cplx* input, cplx* outRaw, int xdim_new, int xdim, int ydim, int slicedim, int bitdim, int spec_slice, int numSlice);
__global__ void DoZeroFilledCUDA(cplx* input, cplx* out, int xdim_new, int xdim, int ydim, int bitdim, int numSlice);
__global__ void cufftShift_2D_kernel(cplx* data, int N, int bit, int numSlice);
kernelConf* GenAutoConf_2D(int N);
__device__ float atomicMaxf(float* address, float val);
__global__ void max_reduce(const float* const d_array, float* d_max, const size_t elements);
__global__ void paralel_devide(float * data, float* max, int size);
void CUDAgetSpecificSliceData(cplx* input, cplx* outRaw, int spec_slice, int spec_bit, const int* dim, int xdim_new, int ydim_new, int numSlice, int numThread);
void CUDAzeroFilled(cplx* d_inputOneSlice, cplx *d_inputOneSliceNew, int xdim, int ydim, int bitdim, int xdim_new, int ydim_new, int numSlice, int numThread);
void dataScalingOnDevice(float* d_data, int size, int numThread);
int DoRECONSCUDAOperation( cplx* DataCufftOneSlice, float* out, int *dim, int numSlice, float* times, int numThread);
int DoRSSCUDAOperation(cplx* data, float* out, const int* dim,  int spec_slice, int numSlice, int numThread);
int main(int argc, char* argv[])
{
	if(argc < 6)
	{
		cout << "<Usage> <path/filename> <Start_slice> <Number Slices> <New X dimension (Zero Filled Dimension)> <Number Thread (32 or 512>" << endl;
		return 1;
	}
	clock_t GPU_start1, GPU_end1;
	char *filename = (char*)malloc(100);
	int *dim = (int*)malloc(sizeof(int)*4);
	int *Newdim = (int*)malloc(sizeof(int)*4);
	float *timesRecons = (float*)malloc(sizeof(float)*4);
	sprintf(filename,"%s",argv[1]);
	int slice = atoi(argv[2]);
	int numSlice = atoi(argv[3]);
	int new_x = atoi(argv[4]);
	int numThread = atoi(argv[5]);
	int bit = 8;
	if (new_x < 512)
	{
		cout << "ZIP Dimension Less than 512" << endl;
		new_x = 512;
	}
	else
	{
		int x_comp = 1;
		while(x_comp < new_x)
		{
			x_comp <<= 1;
		}
		if(x_comp > new_x)
		{
			cout << "Zero Filled Dimension Must be Power of TWO" << endl;
			return 1;
		}
	}
	hipDeviceProp_t prop;
	checkCudaErrors(hipGetDeviceProperties(&prop, 0));
	readHeaderFile(filename, dim);
	int sizeCfl = 1;
	for(int i = 0; i < 4; i++)
	{
		sizeCfl *= dim[i];
		Newdim[i] = dim[i];
	}
	cplx *data ;
	data = (cplx*)malloc(sizeof(cplx)*sizeCfl);
	int ret = readCflFile(filename, data);
	if(ret == 1)
	{
		cout << "Error on Reading CFL File" << endl;
		return 1;
	}
	int xdim = dim[0];
	int ydim = dim[1];
	int bitdim = dim[3];
	int xdim_new = new_x;
	int ydim_new = xdim_new;
	int sizeImageNew = xdim_new*ydim_new;
	int sizeImage = xdim*ydim;
	int sizeImageNewSlice = sizeImageNew* bitdim *numSlice;
	int sizeImageNewSliceRaw = sizeImage* bitdim *numSlice;
	size_t nBytes_C = sizeof(cplx)*sizeImageNewSlice;
	size_t nBytes_CRaw = sizeof(cplx)*sizeImageNewSliceRaw;
	size_t nBytes_F = sizeof(float)*sizeImageNew*numSlice;
	size_t nBytes_FRSS = sizeof(float)*sizeImage*numSlice;
	cplx* dataManySliceRaw ;
	cplx* d_data;
	checkCudaErrors(hipMalloc((void**)&dataManySliceRaw, nBytes_CRaw));
	checkCudaErrors(hipMalloc((void**)&d_data, sizeof(cplx)*sizeCfl));
	checkCudaErrors(hipMemcpy(d_data, data, sizeof(cplx)*sizeCfl, hipMemcpyHostToDevice));
	free(data);
	GPU_start1  = clock();
	CUDAgetSpecificSliceData(d_data, dataManySliceRaw, slice, bit, dim, xdim_new, ydim_new, numSlice, numThread);
	checkCudaErrors(hipFree(d_data));
	cplx* dataManySlice ;
	checkCudaErrors(hipMalloc((void**)&dataManySlice, nBytes_C));
	GPU_end1 = clock();
	float GPUTimer_getspec = (float)(GPU_end1 - GPU_start1)/CLOCKS_PER_SEC;
	GPU_start1  = clock();
	CUDAzeroFilled(dataManySliceRaw, dataManySlice, xdim, ydim, bitdim, xdim_new, ydim_new, numSlice, numThread);
	Newdim[0] = xdim_new;
	Newdim[1] = ydim_new;
	xdim = dim[0];
	ydim = dim[1];
	bitdim = dim[3];
	GPU_end1 = clock();
	float GPUTimer_zip = (float)(GPU_end1 - GPU_start1)/CLOCKS_PER_SEC;
	GPU_start1  = clock();
	float* dataCUFFT_F ;
	dataCUFFT_F = (float*)malloc(nBytes_F);
	int retCUFFT = DoRECONSCUDAOperation(dataManySlice, dataCUFFT_F, Newdim, numSlice, timesRecons, numThread);
	if(retCUFFT == 1)
	{
		cout << "DoRECONSCUDAOperation is FAILED" << endl;
		return 1;
	}
	GPU_end1 = clock();
	float GPUTimer_fft = timesRecons[0]+timesRecons[1]+timesRecons[2]+timesRecons[3];
	GPU_start1  = clock();
	float *rss;
	rss = (float*)malloc(nBytes_FRSS);
	int retRSS = DoRSSCUDAOperation(dataManySliceRaw, rss, dim, slice, numSlice, numThread);
	if(retRSS == 1)
	{
		cout << "RSS Operation is FAILED" << endl;
		return 1;
	}
	GPU_end1 = clock();
	float GPUTimer_RSSRaw = (float)(GPU_end1 - GPU_start1)/CLOCKS_PER_SEC;
	float GPUTimer = GPUTimer_getspec + GPUTimer_zip + GPUTimer_fft + GPUTimer_RSSRaw;
	cout << "==========================================================================================================" << endl;
	cout << "<path/filename> <Start Slice> <Number Slice Per Operatio> <New X dimension (Zero Filled Dimension)> <Number Thread (32 or 512>" << endl;
	cout << "<kspace> " << std::to_string(slice) << " " << std::to_string(numSlice) << " " << std::to_string(new_x) << " " << std::to_string(numThread)<< endl;
	cout << " " << endl;
	cout << "Timing - Get Specific Slice Data on GPU Processor : " << std::to_string(GPUTimer_getspec*1000) << " ms" << endl;
	cout << "Timing - ZIP OPERATION on GPU Processor : " << std::to_string(GPUTimer_zip*1000) << " ms" << endl;
	cout << "Timing - FFT + FFTSHIFT etc for Recons Image on GPU Processor : " << std::to_string(GPUTimer_fft*1000) << " ms" << endl;
	cout << "         Timing - FFT on GPU Processor : " << std::to_string(timesRecons[0]*1000) << " ms" << endl;
	cout << "         Timing - FFTSHIFT on GPU Processor : " << std::to_string(timesRecons[1]*1000) << " ms" << endl;
	cout << "         Timing - RSS on GPU Processor : " << std::to_string(timesRecons[2]*1000) << " ms" << endl;
	cout << "         Timing - Data Scaling on GPU Processor : " << std::to_string(timesRecons[3]*1000) << " ms" << endl;
	cout << "Timing - RSS Operation for Raw Image on GPU Processor : " << std::to_string(GPUTimer_RSSRaw*1000) << " ms" << endl;
	cout << "CUDA I GPU Timing using CPU Timer : " << std::to_string(GPUTimer*1000) << " ms" << endl;
	cout << "==========================================================================================================" << endl;
	/*int retCV = DoDisplayImage2CV(rss, dataCUFFT_F, xdim, ydim, slice, new_x, filename, numSlice);
	if(retCV == 1)
	{
		cout << "Display Image is Failed" << endl;
		return 1;
	}*/
	free(filename);
	free(Newdim);
	free(dim);
	free(timesRecons);
	checkCudaErrors(hipFree(dataManySliceRaw));
	checkCudaErrors(hipFree(dataManySlice));
	free(dataCUFFT_F);
	free(rss);
	checkCudaErrors(hipDeviceReset());
	return 0;
}

void readHeaderFile(char *filen, int * dims)
{
	char path[20];
	sprintf(path,"%s.hdr",filen);
	FILE *myFile;
	myFile = fopen(path,"r");
	string line;
	streampos size;
	
	fseek(myFile, 13, SEEK_SET);
	for(int i = 0; i < 4; i++)
	{
		fscanf(myFile,"%d",&dims[i]);
	}
}
int readCflFile(char *filen, cplx* data)
{
	streampos size;
	char path[20];
	sprintf(path,"%s.cfl",filen);
	ifstream file(path, ios::in | ios::binary | ios::ate);
	if(file.is_open())
	{
		size = file.tellg();
		if(size)
		{
			// cout << "Read CFL file Success" << endl;
		}
		else
		{
			cout << "Error Allocating Memory" << endl;
			return 1;
		}
	}
	else
	{
		cout << "Unable to open file";
		return 1; 
	}
	if(file.is_open())
	{
		file.seekg(0, ios::beg);
		file.read((char*)data, size);
		file.close();
	}
	return 0;
}
void CUDAgetSpecificSliceData(cplx* d_input, cplx* d_outRaw, int spec_slice, int spec_bit, const int* dim, int xdim_new, int ydim_new, int numSlice, int numThread )
{
	int xdim = dim[0];
	int ydim = dim[1];
	int slicedim = dim[2];
	int bitdim = dim[3];
	dim3 tpb(xdim, ydim);
	dim3 bpg(1,1);
	if(xdim > numThread)
	{
		tpb.x = numThread;
		tpb.y = numThread;
		bpg.x = ceil(double(xdim)/double(tpb.x));
		bpg.y = ceil(double(xdim)/double(tpb.y));
	}
	DoGetSpecificSliceDataCUDA<<<tpb, bpg>>>(d_input, d_outRaw, xdim_new, xdim, ydim, slicedim, bitdim, spec_slice, numSlice);
	checkCudaErrors(hipDeviceSynchronize());
}
__global__ void DoGetSpecificSliceDataCUDA(cplx* input, cplx* out, int xdim_new, int xdim, int ydim, int slicedim, int bitdim, int spec_slice, int numSlice)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int indexOne = idy * xdim + idx;
	int N = xdim*ydim;
	int offsetBitmask = N*numSlice;
	int offsetSlice = N;
	int sliceP = 0;
	if(idx < xdim && idy < ydim)
	{
		for(int m = 0; m < bitdim; m++)
		{
			sliceP = 0;
			for(int l = spec_slice; l < (spec_slice+numSlice); l++)
			{
				out[indexOne + (sliceP*offsetSlice) + (m*offsetBitmask)] = input[indexOne + (l*N) + (m*slicedim*N)];
				sliceP++;
			}
		}
	}
}
void CUDAzeroFilled(cplx* d_inputOneSlice, cplx *d_inputOneSliceNew, int xdim, int ydim, int bitdim, int xdim_new, int ydim_new, int numSlice, int numThread)
{
	dim3 tpb(xdim, ydim);
	dim3 bpg(1,1);
	if(xdim > numThread)
	{
		tpb.x = numThread;
		tpb.y = numThread;
		bpg.x = ceil(double(xdim)/double(tpb.x));
		bpg.y = ceil(double(xdim)/double(tpb.y));
	}
	DoZeroFilledCUDA<<<tpb, bpg>>>(d_inputOneSlice, d_inputOneSliceNew, xdim_new, xdim, ydim, bitdim, numSlice);
	checkCudaErrors(hipDeviceSynchronize());
}
__global__ void DoZeroFilledCUDA(cplx* input, cplx* out, int xdim_new, int xdim, int ydim, int bitdim, int numSlice)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int indexOne = idy * xdim + idx;
	int N = xdim*ydim;
	int residue = (xdim_new-xdim)/2;
	int sidx = (xdim_new*residue)+residue;
	int offset = 2*residue;
	int offsetPerBitmask = xdim_new*xdim_new;
	int offsetBitmask = N*numSlice;
	int offsetSlice = N;
	int offsetPerSlice = xdim_new*xdim_new;
	if(idx < xdim && idy < ydim)
	{
		for(int m = 0; m < bitdim; m++)
		{
			for(int l = 0; l < numSlice; l++)
			{
				out[sidx + indexOne + (idy*offset) + (l*offsetPerSlice) + (m*offsetPerBitmask)] = input[indexOne + (l*offsetSlice) + (m*offsetBitmask)];
			}
		}
	}
}
int DoRECONSCUDAOperation( cplx* d_data, float* out, int *dim, int numSlice, float* times, int numThread)
{
	clock_t start, end;
	start = clock();
	int xdim = dim[0];
	int ydim = dim[1];
	int bitdim = dim[3];
	int sizeOneImage = xdim*ydim;
	size_t nBytes_C = sizeof(cplx)*sizeOneImage;
	size_t nBytes_F = sizeof(float)*sizeOneImage*numSlice;
	float* d_out;
	cplx* d_temp;
	checkCudaErrors(hipMalloc((void**)&d_out, nBytes_F));
	checkCudaErrors(hipMalloc((void**)&d_temp, nBytes_C));
	int offset = 0;
	hipfftHandle plan;
	int batch = 1;
    int nn[RANK] = {sizeOneImage};
    hipfftPlanMany(&plan, RANK, nn, NULL, 0, 0, NULL, 0, 0, HIPFFT_C2C, batch);
    for(int a = 0; a < numSlice*bitdim; a++)
	{
		offset = a*sizeOneImage;
		checkCudaErrors(hipMemcpy(d_temp, d_data+offset, nBytes_C, hipMemcpyDeviceToDevice));
		hipfftExecC2C(plan, d_temp, d_temp, HIPFFT_BACKWARD);
		checkCudaErrors(hipMemcpy(d_data+offset, d_temp, nBytes_C, hipMemcpyDeviceToDevice));
	}
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_temp));
    end = clock();
    times[0] = (float)(end-start)/CLOCKS_PER_SEC;
    start = clock();
	if(xdim != ydim)
	{
		cout << "X Dimension Must be same with YDIM" << endl;
		return 1;
	}
	else
	{
		const int N = xdim;
        kernelConf* conf = GenAutoConf_2D(N);
        cufftShift_2D_kernel <<< conf->grid, conf->block >>> (d_data, N, bitdim, numSlice); // In-Place
        checkCudaErrors(hipDeviceSynchronize());
	}
	end = clock();
    times[1] = (float)(end-start)/CLOCKS_PER_SEC;
    start = clock();
	dim3 threadsPerBlock(sizeOneImage);
	dim3 blocksPerGrid(1);
	if(sizeOneImage > numThread)
	{
		threadsPerBlock.x = numThread;
		blocksPerGrid.x = ceil(double(sizeOneImage)/double(threadsPerBlock.x));
	}
	DoRSSCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_out, sizeOneImage, xdim, ydim, bitdim, numSlice);
	checkCudaErrors(hipDeviceSynchronize());
	end = clock();
    times[2] = (float)(end-start)/CLOCKS_PER_SEC;
    start = clock();
	int sizeManySliceWOBit = sizeOneImage*numSlice;
	dataScalingOnDevice(d_out, sizeManySliceWOBit, numThread);
	checkCudaErrors(hipMemcpy(out, d_out, nBytes_F, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_out));
	hipfftDestroy(plan);
	end = clock();
    times[3] = (float)(end-start)/CLOCKS_PER_SEC;
	return 0;
}
void dataScalingOnDevice(float* d_data, int size, int numThread)
{
	float *max;
	checkCudaErrors(hipMalloc((void**)&max, 1*sizeof(float)));
	int threadMax = 32;
	size_t sharedMem = 3*(4096*(sizeof(float)));
	dim3 tpb(size);
	dim3 bpg(1);
	if(size > threadMax)
	{
		tpb.x = threadMax;
		bpg.x = ceil(double(size)/double(tpb.x));
	}
	max_reduce<<<bpg, tpb, sharedMem>>>(d_data, max, size);
	if(size > numThread)
	{
		tpb.x = numThread;
		bpg.x = ceil(double(size)/double(tpb.x));
	}
	paralel_devide<<<bpg, tpb>>>(d_data, max, size);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(max));
}
__device__ float atomicMaxf(float* address, float val)
{
    int *address_as_int =(int*)address;
    int old = *address_as_int, assumed;
    while (val > __int_as_float(old)) {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
                        __float_as_int(val));
        }
    return __int_as_float(old);
}
__global__ void max_reduce(const float* const d_array, float* d_max, 
                                              const size_t elements)
{
    extern __shared__ float shared[];
    int tid = threadIdx.x;
    int gid = (blockDim.x * blockIdx.x) + tid;
    shared[tid] = -FLOAT_MAX; 
    while (gid < elements) {
        shared[tid] = max(shared[tid], d_array[gid]);
        gid += gridDim.x*blockDim.x;
        }
    __syncthreads();
    gid = (blockDim.x * blockIdx.x) + tid;  // 1
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) 
    {
        if (tid < s && gid < elements)
            shared[tid] = max(shared[tid], shared[tid + s]);
        __syncthreads();
    }
    if (tid == 0)
      atomicMaxf(d_max, shared[0]);
}
__global__ void paralel_devide(float * data, float* max, int size)
{
	int idx = (blockDim.x*blockIdx.x) + threadIdx.x;
	if(idx < size)
	{
		data[idx] = data[idx]/max[0];
	}
}
__global__ void DoRSSCUDA(cplx* input, float* out, int N, int x, int y, int bit, int numSlice)
{
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	float temp;
	float sum;
	int sizeManySlice = N*numSlice;
	if(tidx < N) 
	{
		for(int j = 0; j < numSlice; j++)
		{
			temp = 0;
			sum = 0;
			for(int i = 0; i < bit; i++)
			{
				temp = hipCabsf(input[tidx+(j*N)+(i*sizeManySlice)]);
				temp = (float)pow((double)temp, (double)2);
				sum += temp;
			}
			sum = (float)sqrt(sum);
			out[tidx+(j*N)] = sum;
		}
	}
}

int DoDisplayImage2CV(float* dataRaw, float* dataRecons, int xdim, int ydim, int spec_slice, int zipdimx, char* filename, int numSlice)
{
	char winNameRaw[100];
	char winNameRecons[100];
	cv::Mat imgRaw;
	cv::Mat imgRecons;
	int oneDimRaw = xdim*ydim;
	int oneDim = zipdimx*zipdimx;
	size_t nBytes_One = sizeof(float)*oneDim;
	size_t nBytes_OneRaw = sizeof(float)*oneDimRaw;
	float* oneImage = (float*)malloc(nBytes_One);
	float* oneImageRaw = (float*)malloc(nBytes_OneRaw);
	int offset = 0;
	int offsetRaw = 0;
	for(int a = 0; a < numSlice; a++)
	{
		offset = a*oneDim;
		offsetRaw = a*oneDimRaw;
		memcpy(oneImage, dataRecons + offset, nBytes_One);
		memcpy(oneImageRaw, dataRaw + offsetRaw, nBytes_OneRaw);
		imgRaw = cv::Mat(xdim, ydim, CV_32F, oneImageRaw);
		imgRecons = cv::Mat(zipdimx, zipdimx, CV_32F, oneImage);
		if(imgRaw.rows == 0 || imgRaw.cols == 0)
			return 1;
		if(imgRecons.rows == 0 || imgRecons.cols == 0)
			return 1;
		sprintf(winNameRecons,"Reconstructed Image on CV - Slice %d",spec_slice+a );
		sprintf(winNameRaw,"Raw Image on CV - Slice %d",spec_slice+a );
		cv::namedWindow(winNameRaw, CV_WINDOW_KEEPRATIO | CV_WINDOW_NORMAL);
		cv::namedWindow(winNameRecons, CV_WINDOW_KEEPRATIO | CV_WINDOW_NORMAL);
		cv::imshow(winNameRaw,imgRaw);
		cv::waitKey(500);
		cv::imshow(winNameRecons,imgRecons);
		cv::waitKey(1500);
	}
	cv::waitKey();
	free(oneImage);
	free(oneImageRaw);
	return 0;
}
kernelConf* GenAutoConf_2D(int N)
{
    kernelConf* autoConf = (kernelConf*) malloc(sizeof(kernelConf));
    int threadsPerBlock_X;
    int threadsPerBlock_Y;
    if (2 <= N && N < 4)
    {
        threadsPerBlock_X = 2;
        threadsPerBlock_Y = 2;
    }
    if (4 <= N && N < 8)
    {
        threadsPerBlock_X = 4;
        threadsPerBlock_Y = 4;
    }
    if (8 <= N && N < 16)
    {
        threadsPerBlock_X = 8;
        threadsPerBlock_Y = 8;
    }
    if (16 <= N && N < 32)
    {
        threadsPerBlock_X = 16;
        threadsPerBlock_Y = 16;
    }
    if (N >= 32)
    {
        threadsPerBlock_X = 32;
        threadsPerBlock_Y = 32;
    }
    autoConf->block = dim3(threadsPerBlock_X, threadsPerBlock_Y, 1);
    autoConf->grid = dim3((N / threadsPerBlock_X), (N / threadsPerBlock_Y), 1);
    return autoConf;
}
__global__
void cufftShift_2D_kernel(cplx* data, int N, int bit, int numSlice)
{
    int sLine = N;
    int sSlice = N * N;
    int sManySlice = N * N*numSlice;
    int sEq1 = (sSlice + sLine) / 2;
    int sEq2 = (sSlice - sLine) / 2;
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;
    int index = (yIndex * N) + xIndex;
    cplx regTemp;
    if (xIndex < N / 2)
    {
        if (yIndex < N / 2)
        {
        	for(int i = 0; i<bit; i++)
        	{
        		for(int j = 0; j < numSlice; j++)
        		{
        			regTemp = data[index + (j*sSlice) + (i*sManySlice)]; // I am add "+ (i*sSlice)"
        			data[index+ (j*sSlice) + (i*sManySlice)] = data[index + sEq1 + (j*sSlice) + (i*sManySlice)]; // I am add "+ (i*sSlice)"
        			data[index + sEq1+ (j*sSlice) + (i*sManySlice)] = regTemp; // I am add "+ (i*sSlice)"
        		}
        	}
        }
    }
    else
    {
        if (yIndex < N / 2)
        {
        	for(int i = 0; i< bit; i++)
        	{
        		for(int j = 0; j < numSlice; j++)
        		{
	        		regTemp = data[index + (j*sSlice) + (i*sManySlice)]; // I am add "+ (i*sSlice)"
	        		data[index + (j*sSlice) + (i*sManySlice)] = data[index + sEq2 + (j*sSlice) + (i*sManySlice)]; // I am add "+ (i*sSlice)"
	        		data[index + sEq2 + (j*sSlice) + (i*sManySlice)] = regTemp; // I am add "+ (i*sSlice)"
	        	}
        	}
        }
    }
}
int DoRSSCUDAOperation(cplx* d_data, float* out, const int* dim,  int spec_slice, int numSlice, int numThread)
{
	int xdim = dim[0];
	int ydim = dim[1];
	int bitdim = dim[3];
	int sizeOneImg = xdim*ydim;
	size_t nBytes_F = sizeof(float)*sizeOneImg*numSlice;
	float* d_out;
	checkCudaErrors(hipMalloc((void**)&d_out, nBytes_F));
	if(xdim != ydim)
	{
		cout << "X Dimension Must be same with YDIM" << endl;
		return 1;
	}
	dim3 threadsPerBlock(sizeOneImg);
	dim3 blocksPerGrid(1);
	if(sizeOneImg > numThread)
	{
		threadsPerBlock.x = numThread;
		blocksPerGrid.x = ceil(double(sizeOneImg)/double(threadsPerBlock.x));
	}
	DoRSSCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_out, sizeOneImg, xdim, ydim, bitdim, numSlice);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(out, d_out, nBytes_F, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_out));
	return 0;
}